#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream> 
#include <time.h>
#include <stdlib.h>
#include <math.h> 
#include <random>
#include <chrono> 

template<typename T>
void fillArrayWithRandomData(T* arr, int arrayLength, const T* charTable, int charTableLength) {
    auto seed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
    std::mt19937 gen(seed);
    std::uniform_int_distribution<int> dist(0, charTableLength - 1);

    for (int i = 0; i < arrayLength; ++i) {
        int randomIndex = dist(gen);
        arr[i] = charTable[randomIndex];
    }
}

void showCharArray(char* arr, int len) {
    for (int i = 0; i < len; i++) {
        printf("%c", arr[i]);
    }
    printf("\n");
}

template<typename T>
class cuArray {
private:

    __host__ __device__ int __getOffset(int* ptr, int r, int n) {
        return n + r;
    }

    template<typename ...Args>
    __host__ __device__
        int __getOffset(int* ptr, int r, int n, Args ... args) {
        return this->__getOffset(ptr + 1, (r + n) * (*(ptr + 1)), args...);
    }

protected:
    T* h_arr = nullptr;           // Host array pointer
    T* d_arr = nullptr;           // Device array pointer

    int arrSize = 1;              // Size of the array
    int shapeSize = 0;            // Number of dimensions of the array
    int* h_shape = nullptr;       // Host shape array pointer
    int* d_shape = nullptr;       // Device shape array pointer

    // L1, L2, ... 
    // c1, c2, ...
    template<typename ...Args>
    int h_getOffsetIndex(Args ... args) {
        return this->__getOffset(this->h_shape, 0, args...);
    }

    template<typename ...Args>
    __device__
        int d_getOffsetIndex(Args ... args) {
        return this->__getOffset(this->d_shape, 0, args...);
    }

    template<typename ...Args>
    T h_getItem(Args... args) {
        return this->h_getItemOffset(this->h_getOffsetIndex(args...));

    }

    template<typename ...Args>
    void h_setItem(T item, Args... args) {
        this->h_setItemOffset(this->h_getOffsetIndex(args...), item);
    }

    template<typename ...Args>
    __device__
        T d_getItem(Args... args) {
        return this->d_getItemOffset(this->d_getOffsetIndex(args...));
    }

    template<typename ...Args>
    __device__
        void d_setItem(T item, Args... args) { 
        this->d_setItemOffset(this->d_getOffsetIndex(args...), item);
    }

    // Initialize the shape of the array
    virtual void initShape(int* shape, int size) {
        this->shapeSize = size;   // Set the number of dimensions
        this->h_shape = new int[size];  // Allocate memory for the shape array
        for (int i = 0; i < size; i++) {
            this->h_shape[i] = shape[i];  // Copy the shape data
        }
        this->calculateArrSize(); // Calculate the total size of the array
        
        this->allocateDeviceShapeArray();
        this->copyShapeToDevice();
    }


    // Calculate the total size of the array
    void calculateArrSize() {
        for (int i = 0; i < this->shapeSize; i++) {
            this->arrSize *= this->h_shape[i];  // Multiply the dimensions to get the total number of elements
        }
    }

public:


    // cuArray constructor: does not modify external data
    cuArray() {};

    // Free allocated memory
    virtual void freeData() {
        if (this->d_arr != nullptr) {
            hipFree(this->d_arr);  // Free device array memory
            this->d_arr = nullptr;
        }
        if (this->d_shape != nullptr) {
            hipFree(this->d_shape);  // Free device shape array memory
            this->d_shape = nullptr;
        }
        if (this->h_shape != nullptr) {
            delete[] this->h_shape;  // Free host shape array memory
            this->h_shape = nullptr;
        }
    }

    // Set the host data
    virtual void setData(T* data) {
        this->h_arr = data;  // Assign the provided data pointer to the host array pointer
    }

    // Get the item at a given offset (host side)
    T h_getItemOffset(int offset) {
        return this->h_arr[offset];
    }

    // Get the item at a given offset (device side)
    __device__ T d_getItemOffset(int offset) {
        return this->d_arr[offset];
    }

    // Set the item at a given offset (host side)
    void h_setItemOffset(int offset, T item) {
        this->h_arr[offset] = item;
    }

    // Set the item at a given offset (device side)
    __device__ void d_setItemOffset(int offset, T item) { 
        this->d_arr[offset] = item;
    }

    // Display the array contents (host side)
    virtual void showArray() {
        if (this->h_arr == nullptr) {
            std::cout << "Array is empty" << std::endl;
            return;
        }

        std::cout << "Array data:" << std::endl;
        for (int i = 0; i < this->arrSize; ++i) {
            std::cout << this->h_arr[i] << " ";
        }
        std::cout << std::endl;
    }

    // Display the array shape (host side)
    void showShape() {
        std::cout << "The size of array: " << (this->arrSize * sizeof(T)) << " byte(s)" << std::endl;
        std::cout << "Shape: [";
        for (int i = 0; i < this->shapeSize; i++) {
            std::cout << this->h_shape[i] << ",";
        }
        std::cout << "]" << std::endl;
    }

    // Allocate memory for the device array
    void allocateDeviceArray() {
        if (this->d_arr == nullptr) {
            hipMalloc((void**)&this->d_arr, sizeof(T) * this->arrSize);
        }
    }

    // Copy data to the device array
    void copyDataToDevice() {
        hipMemcpy(this->d_arr, this->h_arr, sizeof(T) * this->arrSize, hipMemcpyHostToDevice);
    }

    // Allocate memory for the device shape array
    void allocateDeviceShapeArray() {
        if (this->d_shape == nullptr) {
            hipMalloc((void**)&this->d_shape, sizeof(int) * this->shapeSize);
        }
    } 

    // Copy shape data to the device array
    void copyShapeToDevice() {
        hipMemcpy(this->d_shape, this->h_shape, sizeof(int) * this->shapeSize, hipMemcpyHostToDevice);
    }


    // Transfer data from host to device
    virtual void toDevice() {
        this->allocateDeviceArray();
        if(this->h_arr != nullptr)
        {
            this->copyDataToDevice();
        }
    }

    // Transfer data from device to host
    virtual void toHost() {
        hipMemcpy(this->h_arr, this->d_arr, sizeof(T) * this->arrSize, hipMemcpyDeviceToHost); // Copy data to the host array
    }

    void toHost(T* h_ptr) {
        hipMemcpy(h_ptr, this->d_arr, sizeof(T) * this->arrSize, hipMemcpyDeviceToHost); // Copy data to the host array
    }
};


class SeqPair : public cuArray<char> {
public:
    SeqPair(int batchnum, int seqlen) {
        int shape[3] = { batchnum, 2, seqlen };
        this->initShape(shape, 3);
    }

    char h_getItem(int batchId, int seqId, int charId) {
        return cuArray<char>::h_getItem(batchId, seqId, charId);
    }

    void h_setItem(int batchId, int seqId, int charId, char item) {
        cuArray<char>::h_setItem(item, batchId, seqId, charId);
    }

    __device__ char d_getItem(int batchId, int seqId, int charId) {
        return cuArray<char>::d_getItem(batchId, seqId, charId);
        
    }

    __device__ void d_setItem(int batchId, int seqId, int charId, char item) {
        cuArray<char>::d_setItem(item, batchId, seqId, charId);
    }

};

class DpMatrix : public cuArray<int> {
public:
    DpMatrix(int batchNum, int rowNum, int colNum) {
        int shape[3] = { batchNum, rowNum, colNum };
        this->initShape(shape, 3);

    }

    int h_getItem(int batchId, int rowId, int colId) {
        return cuArray<int>::h_getItem(batchId, rowId, colId);
    }

    void h_setItem(int batchId, int rowId, int colId, int item) {
        cuArray<int>::h_setItem(item, batchId, rowId, colId);
    }

    __device__ int d_getItem(int batchId, int rowId, int colId) {
        
        return cuArray<int>::d_getItem(batchId, rowId, colId);
    }

    __device__ void d_setItem(int batchId, int rowId, int colId, int item) {
        cuArray<int>::d_setItem(item, batchId, rowId, colId);
    }
     

    __device__ void debug() {
        printf("debug\n");
        //cuArray<int>::d_getOffsetIndex(0, 1, 2);
        printf("xx%d\n", this->d_getOffsetIndex(0,1,3));
    }
};

class cuResult : public cuArray<int> {
public:
    cuResult(int batchNum) {
        int shape[2] = { batchNum,  };
        this->initShape(shape, 1);
    }

    int h_getItem(int batchId) {
        return cuArray<int>::h_getItem(batchId );
    }

    void h_setItem(int batchId, int item) {
        cuArray<int>::h_setItem(item, batchId );
    }

    __device__ int d_getItem(int batchId) {
        return cuArray<int>::d_getItem(batchId);
    }

    __device__ void d_setItem(int batchId, int item) {
        cuArray<int>::d_setItem(item, batchId);
    }
     
};

__global__ void levenshtein(SeqPair sp, DpMatrix dpmat, cuResult res, int len) {
    int t_id = threadIdx.x + blockDim.x * blockIdx.x;
    int m;
 
    for (int i = 0; i < len + 1; i++) {
        dpmat.d_setItem(t_id, i, 0, i);
    }
    for (int i = 0; i < len + 1; i++) {
        dpmat.d_setItem(t_id, 0, i, i);
    }

    for (int i = 1; i < len + 1; i++) {
        for (int j = 1; j < len + 1; j++) {
           if (sp.d_getItem(t_id, 0, i - 1) == sp.d_getItem(t_id, 1, j - 1)) {
                dpmat.d_setItem(t_id, i, j, dpmat.d_getItem(t_id, i - 1, j - 1));
            }
            else {
                m = min(dpmat.d_getItem(t_id, i, j - 1), min(dpmat.d_getItem(t_id, i - 1, j), dpmat.d_getItem(t_id, i - 1, j - 1))) + 1;
                dpmat.d_setItem(t_id, i, j, m);
            }
        }
    }
    res.d_setItem(t_id, dpmat.d_getItem(t_id, len, len));
}

int levenshteinDistance(const char* s1, const char* s2, int len) {
    int** dp = (int**)malloc((len + 1) * sizeof(int*));
    int cost;
    for (int i = 0; i <= len; i++) {
        dp[i] = (int*)malloc((len + 1) * sizeof(int));
    }

    for (int i = 0; i <= len; i++) {
        dp[i][0] = i;
        dp[0][i] = i;
    }

    for (int i = 1; i <= len; i++) {
        for (int j = 1; j <= len; j++) { 
            if (s1[i - 1] == s2[j - 1]) {
                cost = 0;
            }
            else {
                cost = 1;
            }
            dp[i][j] = min(dp[i - 1][j] + 1,  
                min(dp[i][j - 1] + 1,
                    dp[i - 1][j - 1] + cost)); // 替换
        }
    }

    int result = dp[len][len];

    for (int i = 0; i <= len; i++) {
        free(dp[i]);
    }
    free(dp);

    return result;
}
int main() {
    int b_num = 1024 * 4;
    int seq_len = 32;
    int blockNum = 128;
    int N = 2 * b_num * seq_len;
    char* strs = new char[N];
    char table[4] = { 'A','G','C','T' };
    fillArrayWithRandomData(strs, N, table, 4);
    SeqPair sp = SeqPair(b_num, seq_len);
    DpMatrix dpmat = DpMatrix(b_num, seq_len + 1, seq_len + 1);
    cuResult res = cuResult(b_num);
    int* ans = new int[b_num];
    ans[0] = 2;
    sp.setData(strs);
    std::cout << "block num: " << blockNum << " block size: " << b_num / blockNum << std::endl;
    clock_t start = clock();
    sp.toDevice();
    dpmat.toDevice();
    res.toDevice();
    levenshtein << <blockNum, b_num / blockNum >> > (sp, dpmat, res, seq_len);
    res.toHost(ans);
    clock_t end = clock();
    std::cout <<"gpu: " << end - start << std::endl;
    int a;
    start = clock();
    for (int i = 0; i < b_num; i++) {
        a = levenshteinDistance(strs + 2 * i * seq_len, strs + 2 * i * seq_len + seq_len, seq_len);
        //std::cout << ans[i]<<" "<< a <<" "<<(a == ans[i]) << std::endl;
        if (a != ans[i]) {
            std::cout << "error" << std::endl;
        }
    }
    end = clock();
    std::cout<<"cpu: " << end - start << std::endl;
    delete[] ans;
    delete[] strs;
    sp.freeData();
    res.freeData();
    dpmat.freeData();
    return 0;
}
